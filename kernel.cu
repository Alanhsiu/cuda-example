#include <hip/hip_runtime.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}
