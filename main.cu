#include <iostream>
#include <hip/hip_runtime.h>

// Declare the kernel function
__global__ void addKernel(int* c, const int* a, const int* b, int size);

void addArrays(const int* a, const int* b, int* c, int size) {
    int *d_a, *d_b, *d_c;

    // Allocate GPU memory
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_b, size);

    // Copy result back to the host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    const int arraySize = 5;
    int a[arraySize] = {1, 2, 3, 4, 5};
    int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    addArrays(a, b, c, arraySize);

    std::cout << "Result: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
